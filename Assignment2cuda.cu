
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
using namespace std;

__global__ void addKernel(int *d_d, const int *d_a, const int *d_b, const int *d_c){
    int i = threadIdx.x;
    d_d[i] = d_a[i] + d_b[i] + d_c[i];
}

int main(void){
    const int SIZE = 5;
    int a[SIZE], b[SIZE], c[SIZE], d[SIZE];
    int *d_a, *d_b, *d_c, *d_d;
    
    srand((unsigned int)time(NULL));
    for(int i = 0; i < SIZE; i++){
        a[i] = rand() % 100;
        b[i] = rand() % 100;
        c[i] = rand() % 100;
    }

    hipMalloc(&d_a, SIZE * sizeof(int));
    hipMalloc(&d_b, SIZE * sizeof(int));
    hipMalloc(&d_c, SIZE * sizeof(int));
    hipMalloc(&d_d, SIZE * sizeof(int));
    
    hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);
    
    addKernel<<< 1, SIZE >>> (d_d, d_a, d_b, d_c);

    hipMemcpy(d, d_d, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    for(int i = 0; i < SIZE; i++)
        cout << i << " : " << a[i] << " + "  << b[i] << " + " << c[i] << " = " << d[i] << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);

    return 0;
}